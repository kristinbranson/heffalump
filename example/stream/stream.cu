#include "hip/hip_runtime.h"
#include <windows.h>
#include <new>
#include <stdexcept>
#include <hip/hip_runtime.h>

#define ERR(...) logger(1,__FILE__,__LINE__,__FUNCTION__,__VA_ARGS__)
#define CHECK(e) do{if(!(e)){ERR("Expression evaluated to false:\n\t%s",#e); throw std::runtime_error("check failed");}}while(0)
#define CUTRY(e) do{auto ecode=(e); if(ecode!=hipSuccess) {ERR("CUDA: %s",hipGetErrorString(ecode)); throw std::runtime_error(hipGetErrorString(ecode));}} while(0)

#define NELEM (1<<24)
#define NSTREAM (4)
#define NREPS (1<<5)

#define LOG(...) logger(0,__FILE__,__LINE__,__FUNCTION__,__VA_ARGS__) 

static void logger(int is_error,const char *file,int line,const char* function,const char *fmt,...) {
    char buf1[1024]={0},buf2[1024]={0};
    va_list ap;
    va_start(ap,fmt);
    vsprintf(buf1,fmt,ap);
    va_end(ap);
#if 1
    sprintf(buf2,"%s(%d): %s()\n\t - %s\n",file,line,function,buf1);
#else
    sprintf(buf2,"%s\n",buf1);
#endif
    OutputDebugStringA(buf2);
}

__global__
void unaryop(float * __restrict__ out,const float * __restrict__ a) {
    auto i=threadIdx.x+blockIdx.x*blockDim.x;
    out[i]=sqrtf(a[i]);
}

__global__
void binaryop(float * __restrict__ out,const float * __restrict__ a, const float * __restrict__ b) {
    auto i=threadIdx.x+blockIdx.x*blockDim.x;
    out[i]=a[i]*b[i];
}

int WinMain(HINSTANCE hinst,HINSTANCE hprev, LPSTR cmd,int show) {
    auto a=new float[NELEM];
    auto b=new float[NELEM];
    struct {
        float *a,*b,*ab,*a2,*b2,*ab2;
    } dev[2];



    

    try { 
        CUTRY(hipSetDevice(1));
        {
            hipDeviceProp_t prop;
            int id;
            CUTRY(hipGetDevice(&id));
            CUTRY(hipGetDeviceProperties(&prop,id));
            LOG("CUDA: %s",prop.name);
        }

        for(auto j=0;j<2;++j) {
            CUTRY(hipMalloc(&dev[j].a,sizeof(*a)*NELEM));
            CUTRY(hipMalloc(&dev[j].b,sizeof(*b)*NELEM));
            CUTRY(hipMalloc(&dev[j].ab,sizeof(*b)*NELEM));
        }

        hipStream_t stream[2][NSTREAM];
        for(auto j=0;j<2;++j)
        for(auto i=0;i<NSTREAM;++i)
            CUTRY(hipStreamCreate(&stream[j][i]));

        hipEvent_t start,stop;
        CUTRY(hipEventCreate(&start));
        CUTRY(hipEventCreate(&stop));

        struct {
            hipEvent_t a_uploaded,b_uploaded,ab_done,result_downloaded;
        } es[2];
        
        for(auto i=0;i<2;++i) {
            CUTRY(hipEventCreate(&es[i].a_uploaded,hipEventDisableTiming));
            CUTRY(hipEventCreate(&es[i].b_uploaded,hipEventDisableTiming));
            CUTRY(hipEventCreate(&es[i].ab_done,hipEventDisableTiming));
            CUTRY(hipEventCreate(&es[i].result_downloaded,hipEventDisableTiming));
        }
    
        LOG("Starting");

        hipEventRecord(start,stream[0][0]);
        for(auto i=0;i<NREPS;++i) {
            int j=i%2;

            CUTRY(hipMemcpyAsync(dev[j].a,a,sizeof(*a)*NELEM,hipMemcpyHostToDevice,stream[j][0]));
            CUTRY(hipMemcpyAsync(dev[j].b,b,sizeof(*a)*NELEM,hipMemcpyHostToDevice,stream[j][0]));
            CUTRY(hipStreamWaitEvent(stream[j][0],es[j].result_downloaded,0));
            binaryop<<<NELEM/1024,1024,0,stream[j][0]>>>(dev[j].ab,dev[j].a,dev[j].b);
            CUTRY(hipMemcpyAsync(a,dev[j].ab,sizeof(*a)*NELEM,hipMemcpyDeviceToHost,stream[j][0]));
            CUTRY(hipEventRecord(es[j].result_downloaded,stream[j][0]));
            
        }
        hipEventRecord(stop,stream[1][0]);
        CUTRY(hipStreamSynchronize(stream[0][0]));
        CUTRY(hipStreamSynchronize(stream[1][0]));

        {
            float ms;
            CUTRY(hipEventElapsedTime(&ms,start,stop));
            LOG("Elapsed: %f ms",ms);
        }

        LOG("All Done");

        // Cleanup

        for(auto i=0;i<2;++i) {
            CUTRY(hipEventDestroy(es[i].a_uploaded));
            CUTRY(hipEventDestroy(es[i].b_uploaded));
            CUTRY(hipEventDestroy(es[i].ab_done));
            CUTRY(hipEventDestroy(es[i].result_downloaded));
        }

        for(auto j=0;j<2;++j)
        for(auto i=0;i<NSTREAM;++i)
            hipStreamDestroy(stream[j][i]);
       
//
//        CUTRY(hipFree(dev.a));
//        CUTRY(hipFree(dev.b));
        delete [] a;
        delete [] b;
    return 0;
    } catch(const std::runtime_error &e) {
        ERR("ERROR: %s",e.what());
        return 1;
    }
}
