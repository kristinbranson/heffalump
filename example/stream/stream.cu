#include "hip/hip_runtime.h"
#include <windows.h>
#include <new>
#include <stdexcept>
#include <hip/hip_runtime.h>

#define ERR(...) logger(1,__FILE__,__LINE__,__FUNCTION__,__VA_ARGS__)
#define CHECK(e) do{if(!(e)){ERR("Expression evaluated to false:\n\t%s",#e); throw std::runtime_error("check failed");}}while(0)
#define CUTRY(e) do{auto ecode=(e); if(ecode!=hipSuccess) {ERR("CUDA: %s",hipGetErrorString(ecode)); throw std::runtime_error(hipGetErrorString(ecode));}} while(0)

#define NELEM (1<<24)
#define NSTREAM (2)
#define NREPS (1<<5)

#define LOG(...) logger(0,__FILE__,__LINE__,__FUNCTION__,__VA_ARGS__) 

static void logger(int is_error,const char *file,int line,const char* function,const char *fmt,...) {
    char buf1[1024]={0},buf2[1024]={0};
    va_list ap;
    va_start(ap,fmt);
    vsprintf(buf1,fmt,ap);
    va_end(ap);
#if 1
    sprintf(buf2,"%s(%d): %s()\n\t - %s\n",file,line,function,buf1);
#else
    sprintf(buf2,"%s\n",buf1);
#endif
    OutputDebugStringA(buf2);
}

__global__
void unaryop(float * __restrict__ out,const float * __restrict__ a) {
    auto i=threadIdx.x+blockIdx.x*blockDim.x;
    out[i]=a[i]*a[i];
}

__global__
void binaryop(float * __restrict__ out,const float * __restrict__ a, const float * __restrict__ b) {
    auto i=threadIdx.x+blockIdx.x*blockDim.x;
    out[i]=a[i]*b[i];
}

int WinMain(HINSTANCE hinst,HINSTANCE hprev, LPSTR cmd,int show) {
    auto a=new float[NELEM];
    auto b=new float[NELEM];
    struct {
        float *a,*b,*ab,*a2,*b2,*ab2;
    } dev[NSTREAM];


    try { 
        CUTRY(hipSetDevice(0));
        {
            hipDeviceProp_t prop;
            int id;
            CUTRY(hipGetDevice(&id));
            CUTRY(hipGetDeviceProperties(&prop,id));
            LOG("CUDA: %s\n\tAsync engine count: %d\n\tDevice overlap: %s",prop.name,prop.asyncEngineCount,prop.deviceOverlap?"Yes":"No");
        }

        for(auto j=0;j<NSTREAM;++j) {
            CUTRY(hipMalloc(&dev[j].a,sizeof(*a)*NELEM));
            CUTRY(hipMalloc(&dev[j].b,sizeof(*b)*NELEM));
            CUTRY(hipMalloc(&dev[j].ab,sizeof(*b)*NELEM));
            CUTRY(hipMalloc(&dev[j].a2,sizeof(*a)*NELEM));
            CUTRY(hipMalloc(&dev[j].b2,sizeof(*b)*NELEM));
            CUTRY(hipMalloc(&dev[j].ab2,sizeof(*b)*NELEM));
        }

        hipStream_t stream[NSTREAM];
        for(auto i=0;i<NSTREAM;++i)
            CUTRY(hipStreamCreate(&stream[i]));

        hipEvent_t start,stop;
        CUTRY(hipEventCreate(&start));
        CUTRY(hipEventCreate(&stop));

        struct {
            hipEvent_t a_uploaded,b_uploaded,ab_done,result_downloaded;
        } es[NSTREAM];
        
        for(auto i=0;i<NSTREAM;++i) {
            CUTRY(hipEventCreate(&es[i].a_uploaded,hipEventDisableTiming));
            CUTRY(hipEventCreate(&es[i].b_uploaded,hipEventDisableTiming));
            CUTRY(hipEventCreate(&es[i].ab_done,hipEventDisableTiming));
            CUTRY(hipEventCreate(&es[i].result_downloaded,hipEventDisableTiming));
        }
    
        LOG("Starting");

        // Note: All memory commands are processed in the order they are issued,
        // independent of the stream they are enqueued in.

        hipEventRecord(start,stream[0]);

        CUTRY(hipMemcpyAsync(dev[0].a,a,sizeof(*a)*NELEM,hipMemcpyHostToDevice,stream[0]));
        CUTRY(hipMemcpyAsync(dev[0].b,b,sizeof(*a)*NELEM,hipMemcpyHostToDevice,stream[0]));

        for(auto i=0;i<NREPS;++i) {
            auto j=i%NSTREAM;
            auto jn=(i+1)%NSTREAM; // next j

            
            CUTRY(hipStreamWaitEvent(stream[j],es[j].result_downloaded,0));
            unaryop<<<NELEM/1024,1024,0,stream[j]>>>(dev[j].a2,dev[j].a);
            unaryop<<<NELEM/1024,1024,0,stream[j]>>>(dev[j].b2,dev[j].b);
            binaryop<<<NELEM/1024,1024,0,stream[j]>>>(dev[j].ab,dev[j].a,dev[j].b);
            binaryop<<<NELEM/1024,1024,0,stream[j]>>>(dev[j].ab2,dev[j].a2,dev[j].b2);
            CUTRY(hipMemcpyAsync(dev[jn].a,a,sizeof(*a)*NELEM,hipMemcpyHostToDevice,stream[jn]));
            CUTRY(hipMemcpyAsync(a,dev[j].ab,sizeof(*a)*NELEM,hipMemcpyDeviceToHost,stream[j]));
            CUTRY(hipMemcpyAsync(dev[jn].b,b,sizeof(*a)*NELEM,hipMemcpyHostToDevice,stream[jn]));
            CUTRY(hipEventRecord(es[j].result_downloaded,stream[j]));
            
        }
        hipEventRecord(stop,stream[(NREPS-1)%NSTREAM]);
        for(auto i=0;i<NSTREAM;++i)
            CUTRY(hipStreamSynchronize(stream[i]));

        {
            float ms;
            CUTRY(hipEventElapsedTime(&ms,start,stop));
            LOG("Elapsed: %f ms",ms);
        }

        LOG("All Done");

        // Cleanup (or not)
        return 0;
    } catch(const std::runtime_error &e) {
        ERR("ERROR: %s",e.what());
        return 1;
    }
}
