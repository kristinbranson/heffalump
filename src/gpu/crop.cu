#include "hip/hip_runtime.h"
//   Rutuja Patil <patilr@janelia.hhmi.org>
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0

#include"crop.h"
#include<hip/hip_runtime_api.h>
#include<stdio.h>
#include<assert.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define CEIL(num,den) ((num+den-1)/den)

inline void gpuAssert(hipError_t code, const char *file,
                      int line, int abort=1){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),file, line);
      if (abort) exit(code);
   }
}

__global__ void crop(float *out_x ,float* out_y ,const float *in_x ,const float* in_y ,
                     int loc_x ,int loc_y ,int halfsz ,int npatches ,int w , int h , 
                     int view_flag ,int counter){

    const int idx = threadIdx.x + blockIdx.x*blockDim.x;
    const int idy = threadIdx.y + blockIdx.y*blockDim.y;
    const int x_start = loc_x - halfsz;
    const int y_start = loc_y - halfsz;
    const int x_end = loc_x + halfsz;
    const int y_end = loc_y + halfsz;

    const int locx_id = x_start + idx;
    const int locy_id = y_start + idy;
    const int cropsz = 2*halfsz;
    int height = h;
    int xlim, ylim;

    // set the end limits for the crop 
    if(x_end < w){  
        
        xlim = x_end;  
      
    }else{

        xlim = w;

    }
        
    if(y_end < height){

        ylim = y_end;

    }else{

        ylim = height;

    }
        
    // crop patch
    if(x_start > 0 && y_start > 0){

        if(locx_id < xlim && locy_id < ylim){

            out_x[(counter*cropsz) + idx + (idy*cropsz*npatches)] = in_x[locx_id + locy_id*w];
            out_y[(counter*cropsz) + idx + (idy*cropsz*npatches)] = in_y[locx_id + locy_id*w];

        }else{
               
            out_x[(counter*cropsz) + idx + (idy*cropsz*npatches)] = 0;    
            out_y[(counter*cropsz) + idx + (idy*cropsz*npatches)] = 0;
        }
  
    }else if(locx_id >= 0 && locy_id >=0){

        if(locx_id < xlim && locy_id < ylim){

            out_x[(counter*cropsz) + idx + (idy*cropsz*npatches)] = in_x[locx_id + locy_id*w];
            out_y[(counter*cropsz) + idx + (idy*cropsz*npatches)] = in_y[locx_id + locy_id*w];

        }else{
                   
            out_x[(counter*cropsz) + idx + (idy*cropsz*npatches)] = 0;
            out_y[(counter*cropsz) + idx + (idy*cropsz*npatches)] = 0;
        }
           
    }else{

        out_x[(counter*cropsz) + idx + (idy*cropsz*npatches)] = 0;   
        out_y[(counter*cropsz) + idx + (idy*cropsz*npatches)] = 0;                            
    }
}

struct workspace{  
 
    workspace(struct CropContext *crp){
     
        gpuErrChk(hipMalloc(&out_x ,nbytes_cropsz(crp->halfcropsz,crp->crp_params.npatches)));
        gpuErrChk(hipMalloc(&out_y ,nbytes_cropsz(crp->halfcropsz,crp->crp_params.npatches)));
             
    }

    ~workspace(){    
                
        gpuErrChk(hipFree(out_x));
        gpuErrChk(hipFree(out_y));

    }

    size_t nbytes_cropsz(int halfcropsz,int npatches){
        int cropsz  = 2*halfcropsz;
        return((cropsz*cropsz*npatches)*sizeof(float));  
    }

    size_t result_bytes(int size){
        return(size*sizeof(float));

    }

    void copy_result(const struct CropContext *crp ,void* buf ,size_t size){
        
        int cropsz = 2*crp->halfcropsz;
        float* hout_x = (float*)buf;
        float* hout_y = (float*)(buf + cropsz*cropsz);
        gpuErrChk(hipMemcpyAsync(hout_x ,out_x ,size ,hipMemcpyDeviceToHost));
        gpuErrChk(hipMemcpyAsync(hout_y ,out_y , size ,hipMemcpyDeviceToHost))

    }

    float *out_x;
    float *out_y;
};


void cropPatch(const struct CropContext *self ,const float *in_x ,
               const float *in_y ,int w ,int h){

    if(!self->workspace) return;
    
    int cropsz =2*self->halfcropsz;
    float* out_x = self->out_x;
    float* out_y = self->out_y;
    int side;
   
    dim3 block(32,8);
    dim3 grid(CEIL(cropsz,block.x),CEIL(cropsz,block.y));
   
    // crop for number of side views
    side=1;    
    for(int i = 0;i < self->crp_params.npatches;i++){

        crop<<<grid,block>>>(out_x ,out_y ,in_x ,in_y ,self->crp_params.interest_pnts[2*i], 
                             self->crp_params.interest_pnts[2*i+1] ,
                             self->halfcropsz,self->crp_params.npatches,w,h,side,i);
        hipGetLastError();
    }
   
    hipDeviceSynchronize();

}

// Initialize params for a crop
struct CropContext CropInit(int cellw,int cellh,const struct CropParams params){

    assert(cellw==cellh);
    int halfcropsz = (params.ncells*cellw)/2;
    struct CropContext crp = {0};
    crp.halfcropsz = halfcropsz;
    crp.crp_params = params; 
    workspace *ws = new workspace(&crp);
    crp.workspace = ws;
    crp.out_x = ws->out_x;
    crp.out_y = ws->out_y;
    return crp;
}

//compute the crop
void CropImage(const struct CropContext *self, const float *in_x ,
               const float *in_y ,int width ,int height){

    if(!self->workspace) return;
    cropPatch(self ,in_x ,in_y ,width ,height);

}

//copy the crop output 
void CropOutputCopy(const struct CropContext *self ,void *buf ,size_t sz){
    
    if(!self->workspace) return;
    workspace *ws = (workspace*)self->workspace;
    ws->copy_result(self ,buf ,sz);        

}

// calculate the number of crop output image bytes
size_t CropOutputByteCount(const struct CropContext *self){

    if(!self->workspace) return 0;
    
    int cropsz = self->halfcropsz*2;
    return(((workspace*)self->workspace)->result_bytes(self->crp_params.npatches*cropsz*cropsz));

}


// delete the crop context
void CropTearDown(const struct CropContext *self){

   if(!self->workspace) return;
   workspace *ws = (workspace*)self->workspace;
   delete ws;

}
