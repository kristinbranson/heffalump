#include "../lk.h"
#include <new>
#include <stdexcept>
#include <hip/hip_runtime.h>

#define ERR(L,...) L(1,__FILE__,__LINE__,__FUNCTION__,__VA_ARGS__)
#define CHECK(L,e) do{if(!(e)){ERR(L,"Expression evaluated to false:\n\t%s",#e); throw std::runtime_error("check failed");}}while(0)
#define CUTRY(L,e) do{auto ecode=(e); if(ecode!=hipSuccess) {ERR(L,"CUDA: %s",hipGetErrorString(ecode)); throw std::runtime_error(hipGetErrorString(ecode));}} while(0)


namespace priv {
namespace lk {
namespace gpu {

    using logger_t = void (*)(int is_error,const char *file,int line,const char* function,const char *fmt,...);

    unsigned bytes_per_pixel(enum lk_scalar_type type) {
        const unsigned bpp[]={1,2,4,8,1,2,4,8,4,8};
        return bpp[type];
    }

    struct workspace {
        workspace(logger_t logger, enum lk_scalar_type, unsigned w, unsigned h, unsigned p, const struct lk_parameters& params) 
        : logger(logger)
        , w(w), h(h), pitch(p)
        , params(params)
        {
            CUTRY(logger,hipMalloc(&out,bytesof_output()); 
            CUTRY(logger,hipMemset(input,0,bytesof_input()));
            CUTRY(logger,hipMemset(last,0,bytesof_input()));
            CUTRY(logger,hipMemset(dt,0,w*h*sizeof(float)));

            float *ks[]={self->kernels.derivative,self->kernels.derivative};
            unsigned nks0[]={self->kernels.nder,0};
            unsigned nks1[]={0,self->kernels.nder};
            self->dx=conv_init(logger,w,h,w,ks,nks0);
            self->dy=conv_init(logger,w,h,w,ks,nks1);
        }

        ~workspace() {
            CUTRY(logger,hipFree(last));
            CUTRY(logger,hipFree(out));
        }

        void compute(const float* im) {
            CUTRY(hipMemcpy(input,im,bytesof_input(),hipMemcpyHostToDevice));

            Error:;
        }

        size_t bytesof_input() const {
            return bytes_per_pixel(type)*pitch*h;
        }

        size_t bytesof_output() const {
            return sizeof(float)*w*h*2;
        }

        void copy_last_result(void * buf,size_t nbytes) {
            CUTRY(hipMemcpy(buf,out,n,hipMemcpyDeviceToHost));
        }
    private:
        unsigned w,h,pitch;
        logger_t logger;
        float *out,*last, *input ,*dt;
        struct  {
            struct conv_context dx,dy;        
            float *dt;
        } stage1; // initial computation of gradient in x,y, and t

        struct {
            struct conv_context dx,dy,dt;
        } stage2; // weighting and normalization
        struct lk_parameters params;
    };

}}} // end priv::lk::gpu


using priv::lk::gpu::workspace;

struct lk_context lk_init(
    void (*logger)(int is_error,const char *file,int line,const char* function,const char *fmt,...),
    enum lk_scalar_type type,
    unsigned w,
    unsigned h,
    unsigned pitch,
    const struct lk_parameters params
){
    try {
        workspace *ws=new workspace(logger,&params,w,h,pitch,type);
        struct lk_context self={
            .logger=logger,
            .w=w,
            .h=h,
            .result=ws->out,
            .workspace=ws
        };        
    } catch(const std::runtime_error& e) {
        ERR(logger,"Problem initializing Lucas-Kanade context:\n\t%s",e.what());
    }
Error:
    return self;
}

void lk_teardown(struct lk_context *self) {
    if(!self) return;
    struct workspace* ws=(struct workspace*)self->ws;
    delete ws;
    self->ws=0;
}

void lk(struct lk_context *self,const void *im) {
    struct workspace* ws=(struct workspace*)self->ws;
    ws->compute(im);
}

void* lk_alloc(const struct lk_context *self, void* (*alloc)(size_t nbytes)) {    
    struct workspace* ws=(struct workspace*)self->ws;
    return alloc(ws->bytes_of_output());
}

void  lk_copy(const struct lk_context *self, float *out, size_t nbytes) {
    struct workspace* ws=(struct workspace*)self->ws;
    ws->copy_last_result(out,nbytes);
}